#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "gtable.h"
#include "GPUetc/common/GPUTUPLE.h"

namespace voltdb {

GTable::GTable() {
	database_id_ = 0;
	name_ = NULL;
	block_list_host_ = NULL;
	schema_ = NULL;
	columns_ = 0;
	rows_ = 0;
	block_num_ = 0;
	indexes_ = NULL;
	index_num_ = 0;
}

GTable::GTable(int database_id, char *name, int column_num)
{
	database_id_ = database_id;
	name_ = name;
	block_list_host_ = NULL;
	columns_ = column_num;
	rows_ = 0;
	block_num_ = 0;
	indexes_ = NULL;
	index_num_ = 0;

	block_list_host_ = (GBlock *)malloc(sizeof(GBlock));
	checkCudaErrors(hipMalloc(&block_list_host_[0].data, sizeof(int64_t) * MAX_BLOCK_SIZE));
	checkCudaErrors(hipMalloc(&schema_, sizeof(GColumnInfo) * column_num));
	block_list_host_[0].rows = 0;
	block_list_host_[0].columns = column_num;
	block_list_host_[0].block_size = MAX_BLOCK_SIZE;
}

GTable::GTable(int database_id, char *name, GColumnInfo *schema, int column_num, int rows)
{
	database_id_ = database_id;
	name_ = name;
	block_list_host_ = NULL;
	columns_ = column_num;
	rows_ = rows;
	block_num_ = 0;
	indexes_ = NULL;
	index_num_ = 0;

	block_list_host_ = (GBlock *)malloc(sizeof(GBlock));
	checkCudaErrors(hipMalloc(&block_list_host_[0].data, sizeof(int64_t) * MAX_BLOCK_SIZE));
	checkCudaErrors(hipMalloc(&schema_, sizeof(GColumnInfo) * column_num));
	checkCudaErrors(hipMemcpy(schema_, schema, sizeof(GColumnInfo) * column_num, hipMemcpyHostToDevice));
	block_list_host_[0].rows = rows;
	block_list_host_[0].columns = column_num;
	block_list_host_[0].block_size = MAX_BLOCK_SIZE;
}

void GTable::deleteAllTuples()
{
	for (int i = 0; i < block_num_; i++) {
		checkCudaErrors(hipFree(block_list_host_[i].data));
	}
	free(block_list_host_);
	block_num_ = 0;
	rows_ = 0;
}

void GTable::deleteTuple(int blockId, int tupleId)
{
	if (tupleId < 0 || tupleId > block_list_host_[blockId].rows) {
		printf("Error: tupleId out of range\n");
		return;
	}

	GBlock *target_block = block_list_host_ + blockId;
	int64_t *target_data = target_block->data;

	checkCudaErrors(hipMemcpy(target_data + tupleId * columns_, target_data + (tupleId + 1) * columns_, (target_block->rows - tupleId) * columns_ * sizeof(int64_t), hipMemcpyDeviceToDevice));
	target_block->rows -= 1;
}

void GTable::insertTuple(int64_t *tuple)
{
	int block_id, tuple_id;

	nextFreeTuple(&block_id, &tuple_id);

	int64_t *target_location = block_list_host_[block_id].data + tuple_id * columns_;

	checkCudaErrors(hipMemcpy(target_location, tuple, columns_ * sizeof(int64_t), hipMemcpyHostToDevice));
	block_list_host_[block_id].rows++;
	insertToAllIndexes(block_id, tuple_id);
}

void GTable::insertToAllIndexes(int block_id, int tuple_id)
{
	for (int i = 0; i < index_num_; i++) {
		insertToIndex(block_id, tuple_id, i);
	}
}

void GTable::insertToIndex(int block_id, int tuple_id, int index_id)
{
	return;
}

/* INCOMPLETED */
void GTable::addIndex(int *key_idx, int key_size, GIndexType type)
{
	indexes_ = (GIndex*)realloc(indexes_, sizeof(GIndex) * (index_num_ + 1));
	index_num_++;
}

void GTable::removeIndex()
{
	printf("Error: unsupported operation\n");
	exit(1);
}

void GTable::nextFreeTuple(int *block_id, int *tuple_id)
{
	for (int i = 0; i < block_num_; i++) {
		if (!isBlockFull(i)) {
			*block_id = i;
			*tuple_id = block_list_host_[i].rows;
			return;
		}
	}

	//All current blocks are full, allocate a new one
	GBlock new_block;

	checkCudaErrors(hipMalloc(&new_block.data, MAX_BLOCK_SIZE));
	new_block.columns = columns_;
	new_block.rows = 0;
	new_block.block_size = MAX_BLOCK_SIZE;

	block_list_host_ = (GBlock*)realloc(block_list_host_, sizeof(GBlock) * (block_num_ + 1));
	block_num_++;
}
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "GPUetc/indexes/KeyIndex.h"
#include "GPUetc/indexes/HashIndex.h"

namespace voltdb {
GHashIndex::GHashIndex() {
	key_idx_ = NULL;
	key_size_ = 0;
	bucket_locations_ = NULL;
	bucket_num_ = 0;
	packed_key_ = NULL;

	checkCudaErrors(hipMalloc(&sorted_idx_, sizeof(int) * DEFAULT_PART_SIZE_));	//Default 1024 * 1024 entries
	checkCudaErrors(hipGetLastError());
}

GHashIndex::GHashIndex(int key_num, int key_size)
{
	key_idx_ = NULL;
	key_size_ = key_size;
	bucket_locations_ = NULL;
	bucket_num_ = 0;
	key_num_ = key_num;

	checkCudaErrors(hipMalloc(&sorted_idx_, sizeof(int) * key_num_));
	checkCudaErrors(hipMalloc(&packed_key_, sizeof(uint64_t) * key_num_ * key_size_));
	checkCudaErrors(hipGetLastError());
}


extern "C" __global__ void hashInitialize(GHashIndex table_index, GTable table, int left, int right) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index + left; i <= right; i += stride) {
		GTuple tuple(table, i);

		table_index.insertKeyTupleNoSort(tuple, i);
	}
}


extern "C" __global__ void hashCount(GHashIndex indexes, ulong *hash_count, uint64_t max_buckets)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int tuple_num = indexes.getRows();

	for (int i = index; i < tuple_num; i += stride) {
		GHashIndexKey search_key(indexes, i);
		uint64_t hash = search_key.KeyHasher();
		uint64_t bucket_offset = hash % max_buckets;
		hash_count[bucket_offset * stride + index]++;
	}
}

extern "C" __global__ void bucketsLocate(ulong *hash_count, int *bucket_locations, int bucket_num)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i <= bucket_num; i += stride) {
		bucket_location[i] = hash_count[i * stride];
	}
}

extern "C" __global__ void gHash(GHashIndex indexes, ulong *hash_count, int *sorted_idx, int bucket_num, int start_idx, int end_idx)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i <= end_idx - start_idx + 1; i += stride) {
		GHashIndexKey key(indexes, i + start_idx);
		uint64_t hash = key.KeyHasher();
		uint64_t bucket_offset = hash % bucket_num;
		ulong hash_idx = hash_count[bucket_offset * stride + index];

		sorted_idx[hash_idx] = i;

		hash_count[bucket_offset * stride + index]++;
	}
}


GHashIndex::GHashIndex(GTable table, int *key_idx, int key_size, int bucket_num) {

	key_num_ = table.rows_;
	key_idx_ = key_idx;
	key_size_ = key_size;
	bucket_num_ = bucket_num;

	int block_x = (key_size < BLOCK_SIZE_X) ? key_size : BLOCK_SIZE_X;
	int grid_x = (key_size - 1) / block_x + 1;

	checkCudaErrors(hipMalloc(&bucket_locations_, sizeof(int) * (bucket_num_ + 1)));

	block_x = (key_num_ < BLOCK_SIZE_X) ? key_num_ : BLOCK_SIZE_X;
	grid_x = (key_num_ - 1)/block_x + 1;
	hashInitialize<<<grid_x, block_x>>>(*this, table, 0, key_num_);

	ulong *hash_count;

	checkCudaErrors(hipMalloc(&hash_count, sizeof(ulong) * (bucket_num_ * block_x * grid_x + 1)));

	ulong total;

	hashCount<<<grid_x, block_x>>>(*this, hash_count, bucket_num_);
	GUtilities::ExclusiveScan(hash_count, bucket_num_ * block_x * grid_x + 1, &total);
	checkCudaErrors(hipMalloc(&sorted_idx_, sizeof(int) * DEFAULT_PART_SIZE_));	//Default 1024 * 1024 entries

	int *tmp_bucket_location;

	checkCudaErrors(hipMalloc(&tmp_bucket_locations, sizeof(int) * (bucket_num_ + 1)));

	bucketsLocate<<<grid_x, block_x>>>(*this, tmp_bucket_locations, bucket_num_);
	gHash<<<grid_x, block_x>>>(*this, hash_count, sorted_idx_, bucket_num_, 0, key_num_ - 1);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(hash_count));
}

GHashIndex::~GHashIndex()
{
	if (key_num_ != 0) {
		checkCudaErrors(hipFree(sorted_idx_));
		checkCudaErrors(hipFree(bucket_locations_));
		checkCudaErrors(hipFree(packed_key_));
	}

}

__global__ void hashBucketSearch(GTuple key_tuple, int *bucket_idx)
{
	GHashIndexKey key(key_tuple);

	*bucket_idx = key.KeyHasher();
}

__global__ void hashUpdate(ulong *bucket_location, int bucket_idx, int bucket_num)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < bucket_num - bucket_idx; i += stride) {
		bucket_location[i + bucket_idx + 1]++;
	}
}

void GHashIndex::addEntry(GTuple new_tuple)
{
	int bucket_idx;
	int *dev_bucket_idx;

	checkCudaErrors(hipMalloc(&dev_bucket_idx, sizeof(int)));

	hashBucketSearch<<<1, 1>>>(new_tuple, dev_bucket_idx);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(&bucket_idx, dev_bucket_idx, sizeof(int)));

	ulong copy_location;

	checkCudaErrors(hipMemcpy(&copy_location, bucket_locations_ + bucket_idx + 1, sizeof(ulong), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(sorted_idx_ + copy_location + 1, sorted_idx_ + copy_location, sizeof(int) * (key_num_ - copy_location + 1), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(sorted_idx_ + copy_location, &key_num_, sizeof(int), hipMemcpyHostToDevice));
	key_num_ += 1;
}

/* Add multiple new indexes.
 * New table are already stored in table_ at indexes started from base_idx.
 */
void GHashIndex::addBatchEntry(GTable table, int start_idx, int size) {
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	hashInitialize<<<grid_x, block_x>>>(*this, table, start_idx, start_idx + size - 1);

	ulong *hash_count;

	checkCudaErrors(hipMalloc(&hash_count, sizeof(ulong) * (bucket_num_ * block_x * grid_x + 1)));

	ulong total;

	hashCount<<<grid_x, block_x>>>(*this, hash_count, bucket_num_);
	GUtilities::ExclusiveScan(hash_count, bucket_num_ * block_x * grid_x + 1, &total);

	int *bucket_locations;

	checkCudaErrors(hipMalloc(&bucket_locations, sizeof(int) * (bucket_num_ + 1)));
	bucketsLocate<<<grid_x, block_x>>>(*this, bucket_locations, bucket_num_);
	gHash<<<grid_x, block_x>>>(*this, hash_count, sorted_idx_, bucket_num_, start_idx, start_idx + size - 1);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	quickSort<<<grid_x, block_x>>>(*this, start_idx, start_idx + size - 1);
	checkCudaErrors(hipDeviceSynchronize());

	merge(0, key_num_ - 1, key_num_, key_num_ + size - 1, bucket_locations);
	key_num_ += size;
}

extern "C" __global__ void hashSearchUpper(GHashIndex indexes, int key_left, int key_right, int *output, int *bucket_locations)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i <= key_right - key_left + 1; i += stride) {

		GHashIndexKey key(indexes, i + key_left);

		uint64_t bucket_idx = key.KeyHasher();

		output[i] = bucket_locations[bucket_idx + 1];
	}
}

extern "C" __global__ void hashSearchLower(GHashIndex indexes, int key_left, int key_right, int *output, int *bucket_locations)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i <= key_right - key_left + 1; i += stride) {
		GHashIndexKey key(indexes, i + key_left);

		uint64_t bucket_idx = key.KeyHasher();

		output[i] = bucket_locations[bucket_idx];
	}
}

extern "C" __global__ void constructHashLocation(int *location, int size)
{
	int index = threaIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i += stride) {
		location[i] += i;
	}
}

extern "C" __global__ void hashArrange(int *input, int *output, int *location, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i+= stride) {
		output[location[i]] = input[i];
	}
}
/* Merge new array to the old array
 * Both the new and old arrays are already sorted
 */
void GHashIndex::merge(int old_left, int old_right, int new_left, int new_right, int *bucket_locations) {
	int old_size, new_size;

	old_size = old_right - old_left + 1;
	new_size = new_right - new_left + 1;

	int block_x, grid_x;

	block_x = (new_size < BLOCK_SIZE_X) ? new_size : BLOCK_SIZE_X;
	grid_x = (new_size - 1) / block_x + 1;

	int *write_location;

	checkCudaErrors(hipMalloc(&write_location, (old_size + new_size) * sizeof(int)));
	hashSearchUpper<<<grid_x, block_x>>>(*this, new_left, new_right, write_location + old_size, bucket_locations);
	constructHashLocation<<<grid_x, block_x>>>(write_location + old_size, new_size);

	block_x = (old_size < BLOCK_SIZE_X) ? old_size : BLOCK_SIZE_X;
	grid_x = (old_size - 1)/block_x + 1;

	batchSearchLower<<<grid_x, block_x>>>(*this, old_left, old_right, new_left, new_right, write_location);
	constructHashLocation<<<grid_x, block_x>>>(write_location, old_size);

	block_x = (old_size + new_size < BLOCK_SIZE_X) ? (old_size + new_size) : BLOCK_SIZE_X;
	grid_x = (old_size + new_size - 1)/block_x + 1;

	int *new_sorted_idx;

	checkCudaErrors(hipMalloc(&new_sorted_idx, (old_size + new_size) * sizeof(int)));
	hashArrange<<<grid_x, block_x>>>(sorted_idx_, new_sorted_idx, write_location, old_size + new_size);

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(sorted_idx_));
	sorted_idx_ = new_sorted_idx;
}

int GHashIndex::getBucketNum()
{
	return bucket_num_;
}

}

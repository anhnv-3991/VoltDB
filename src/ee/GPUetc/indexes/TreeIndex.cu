#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "GPUetc/indexes/KeyIndex.h"
#include "GPUetc/indexes/TreeIndex.h"

namespace voltdb {
GTreeIndex::GTreeIndex() {
	key_schema_ = NULL;
	sorted_idx_ = NULL;
	key_idx_ = NULL;
	key_size_ = 0;

	checkCudaErrors(hipMalloc(&sorted_idx_, sizeof(int) * DEFAULT_PART_SIZE_));	//Default 1024 * 1024 entries
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

extern "C" __global__ void setKeySchema(GColumnInfo *key_schema, GColumnInfo *table_schema, int *key_idx, int key_size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < key_size; i += stride) {
		key_schema[i] = table_schema[key_idx[i]];
	}
}

extern "C" __global__ void initialize(GTreeIndex table_index, GTable table, int left, int right) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index + left; i <= right; i += stride) {
		GTuple tuple(table, i);

		table_index.insertKeyTupleNoSort(tuple, i);
	}
}


GTreeIndex::GTreeIndex(GTable table, int *key_idx, int key_size) {

	key_num_ = table.rows_;
	key_idx_ = key_idx;
	key_size_ = key_size;

	int block_x = (key_size < BLOCK_SIZE_X) ? key_size : BLOCK_SIZE_X;
	int grid_x = (key_size - 1) / block_x + 1;

	checkCudaErrors(hipMalloc(&key_schema_, sizeof(GColumnInfo) * key_size_));
	setKeySchema<<<grid_x, block_x>>>(key_schema_, table.schema_, key_idx, key_size);
	checkCudaErrors(hipMalloc(&packed_key, sizeof(int64_t) * key_num_ * key_size_));

	block_x = (key_num_ < BLOCK_SIZE_X) ? key_num_ : BLOCK_SIZE_X;
	grid_x = (key_num_ - 1)/block_x + 1;
	initialize<<<grid_x, block_x>>>(*this, table, 0, key_num_);

	checkCudaErrors(hipMalloc(&sorted_idx_, sizeof(int) * DEFAULT_PART_SIZE_));	//Default 1024 * 1024 entries
	quickSort<<<1, 1>>>(*this, 0, rows_ - 1);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void GTreeIndex::addEntry(GTuple new_tuple) {
	int entry_idx;

	upperBoundSearch<<<1, 1>>>(*this, new_tuple, key_schema_, key_size_);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(sorted_idx_ + entry_idx + 1, sorted_idx_ + entry_idx, sizeof(int) * (rows_ - entry_idx + 1), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(sorted_idx_ + entry_idx, &rows_, sizeof(int), hipMemcpyHostToDevice));
	rows_ += 1;
}

/* Add multiple new indexes.
 * New table are already stored in table_ at indexes started from base_idx.
 *
 * */
void GTreeIndex::addBatchEntry(GTable table, int start_idx, int size) {
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	initialize<<<grid_x, block_x>>>(*this, table, start_idx, start_idx, start_idx + size - 1);
	quickSort<<<grid_x, block_x>>>(*this, start_idx, start_idx + size - 1);
	checkCudaErrors(hipDeviceSynchronize());

	merge(0, key_num_ - 1, key_num_, key_num_ + size - 1);
	key_num_ += size;
}


/* Merge new array to the old array
 * Both the new and old arrays are already sorted
 */

void GTreeIndex::merge(int old_left, int old_right, int new_left, int new_right) {
	int old_size, new_size;

	old_size = old_right - old_left + 1;
	new_size = new_right - new_left + 1;

	int block_x, grid_x;

	block_x = (new_size < BLOCK_SIZE_X) ? new_size : BLOCK_SIZE_X;
	grid_x = (new_size - 1) / block_x + 1;

	int *write_location;

	checkCudaErrors(hipMalloc(&write_location, (old_size + new_size) * sizeof(int)));
	batchSearchUpper<<<grid_x, block_x>>>(*this, new_left, new_right, old_left, old_right, write_location + old_size);
	constructWriteLocation<<<grid_x, block_x>>>(write_location + old_size, new_size);

	block_x = (old_size < BLOCK_SIZE_X) ? old_size : BLOCK_SIZE_X;
	grid_x = (old_size - 1)/block_x + 1;

	batchSearchLower<<<grid_x, block_x>>>(*this, old_left, old_right, new_left, new_right, write_location);
	constructWriteLocation<<<grid_x, block_x>>>(write_location, old_size);

	block_x = (old_size + new_size < BLOCK_SIZE_X) ? (old_size + new_size) : BLOCK_SIZE_X;
	grid_x = (old_size + new_size - 1)/block_x + 1;

	int *new_sorted_idx;

	checkCudaErrors(hipMalloc(&new_sorted_idx, (old_size + new_size) * sizeof(int)));
	rearrange<<<grid_x, block_x>>>(sorted_idx_, new_sorted_idx, write_location, old_size + new_size);

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(sorted_idx_));
	sorted_idx_ = new_sorted_idx;
}

extern "C" {
//Search for the upper bounds of an array of keys

__global__ void batchSearchUpper(GTreeIndex indexes, int key_left, int key_right, int left, int right, int *output) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i <= key_right - key_left + 1; i += stride) {

		GTreeIndexKey key(indexes, i + key_left);

		output[i] = indexes.upperBound(key, left, right);
	}
}

//Search for the lower bounds of an array of keys
__global__ void batchSearchLower(GTreeIndex indexes, int key_left, int key_right, int left, int right, int *output) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i <= key_right - key_left + 1; i += stride) {
		GKeyIndex key(indexes, i + key_left);

		output[i] = indexes.lowerBound(key, left, right);
	}
}

__global__ void constructWriteLocation(int *location, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i += stride) {
		location[i] += i;
	}
}

// Merge the new to the old
__global__ void rearrange(int *input, int *output, int *location, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i+= stride) {
		output[location[i]] = input[i];
	}
}

__global__ void upperBoundSearch(GTreeIndex indexes, GTuple new_tuple) {
	GTreeIndexKey key(new_tuple);
	int rows = indexes.getRows();

	*entry_idx = indexes.upperBound(key, 0, rows - 1);
}

__global__ void lowerBoundSearch(GTreeIndex indexes, GTuple new_tuple, GColumnInfo *key_schema, int key_siz)
{
	GKeyIndex key(tuple, schema, key_idx, key_size);

	*entry_idx = indexes.lowerBound(key, 0, rows - 1);
}

//Quick Sort
__global__ void quickSort(GTreeIndex indexes, int left, int right) {
	if (right <= left)
		return;

	int pivot = (left + right)/2;
	GTreeIndexKey pivot_key(indexes, pivot);
	int left_ptr, right_ptr;


	while (left_ptr <= right_ptr) {
		GTreeIndexKey left_key(indexes, left_ptr);
		GTreeIndexKey right_key(indexes, right_ptr);

		while (GTreeIndexKey::KeyComparator(left_key, pivot_key) < 0) {
			left_ptr++;
			left_key.setKey(indexes, left_ptr);
		}

		while (GKeyIndex::KeyComparator(right_key, pivot_key) > 0) {
			right_ptr--;
			right_key.setKey(indexes, right_ptr);
		}

		if (left_ptr <= right_ptr) {
			indexes.swap(left_ptr, right_ptr);
		}

	}

	if (left < left_ptr) {
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quickSort<<<1, 1, 0, s1>>>(indexes, left, right_ptr);
		hipStreamDestroy(s1);
	}

	if (right > right_ptr) {
		hipStream_t s2;
		hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
		quickSort<<<1, 1, 0, s2>>>(indexes, left_ptr, right);
		hipStreamDestroy(s2);
	}
}
}
}

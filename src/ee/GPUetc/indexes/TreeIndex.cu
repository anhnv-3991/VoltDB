#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "GPUetc/indexes/KeyIndex.h"
#include "GPUetc/indexes/TreeIndex.h"

namespace voltdb {
GTreeIndex::GTreeIndex() {
	key_schema_ = NULL;
	sorted_idx_ = NULL;
	key_idx_ = NULL;
	key_size_ = 0;
	key_num_ = 0;
	packed_key_ = NULL;

	checkCudaErrors(hipMalloc(&sorted_idx_, sizeof(int) * DEFAULT_PART_SIZE_));	//Default 1024 * 1024 entries
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}


GTreeIndex::GTreeIndex(int key_size, int key_num)
{
	key_size_ = key_size;
	key_num_ = key_num;

	checkCudaErrors(hipMalloc(&sorted_idx_, sizeof(int) * key_num_));
	checkCudaErrors(hipMalloc(&packed_key_, sizeof(int64_t) * key_num_ * key_size_));
	checkCudaErrors(hipMalloc(&key_schema_, sizeof(GColumnInfo) * key_size_));
	checkCudaErrors(hipMalloc(&key_idx_, sizeof(int) * key_size_));
}

GTreeIndex::GTreeIndex(int *sorted_idx, int *key_idx, int key_size, int64_t *packed_key, GColumnInfo *key_schema, int key_num)
{
	sorted_idx_ = sorted_idx;
	key_idx_ = key_idx;
	key_size_ = key_size;
	packed_key_ = packed_key;
	key_schema_ = key_schema;
	key_num_ = key_num;
}

extern "C" __global__ void setKeySchema(GColumnInfo *key_schema, GColumnInfo *table_schema, int *key_idx, int key_size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < key_size; i += stride) {
		key_schema[i] = table_schema[key_idx[i]];
	}
}

extern "C" __global__ void initialize(GTreeIndex table_index, int64_t *table, GColumnInfo *schema, int columns, int left, int right)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index + left; i <= right; i += stride) {
		GTuple tuple(table + i * columns, schema, columns);

		table_index.insertKeyTupleNoSort(tuple, i);
	}
}


extern "C" __global__ void mergeSort(GTreeIndex table_index, int left, int right, int *input, int *output, int chunk_size)
{
	int half_size = (right - left)/2 + 1;

	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < half_size; i += blockDim.x * gridDim.x) {
		int left_key_ptr = left + i;
		int right_key_ptr = left_key_ptr + half_size;
		int left_root = left + (i/chunk_size) * chunk_size;
		int right_root = left_root + half_size;
		int left_size = chunk_size;
		int right_size = (right_root + chunk_size > right) ? right_root + chunk_size : right - right_root + 1;
		int base_idx = left + (i / chunk_size) * chunk_size * 2;
		int local_idx = left_key_ptr - left_root;

		GTreeIndexKey left_key = table_index.getKeyAtSortedIndex(left_key_ptr);
		GTreeIndexKey right_key = table_index.getKeyAtSortedIndex(right_key_ptr);
		int new_left_key_ptr = table_index.lowerBound(left_key, right_root, right_root + right_size - 1);
		int new_right_key_ptr = table_index.upperBound(right_key, left_root, left_root + left_size - 1);

		new_left_key_ptr = (new_left_key_ptr != -1) ? (new_left_key_ptr - left_key_ptr + local_idx) : (-1);
		new_right_key_ptr = (new_right_key_ptr != -1) ? (new_right_key_ptr - right_key_ptr + local_idx) : (-1);

		output[new_left_key_ptr + base_idx] = input[left_key_ptr];
		output[new_right_key_ptr + base_idx] = input[right_key_ptr];
	}
}

void GTreeIndex::createIndex(int64_t *table, GColumnInfo *schema, int rows, int columns)
{
	key_num_ = rows;

	int block_x = (key_num_ < BLOCK_SIZE_X) ? key_num_ : BLOCK_SIZE_X;
	int grid_x = (key_num_ - 1) / block_x + 1;

	setKeySchema<<<grid_x, block_x>>>(key_schema_, schema, key_idx_, key_size_);

	GTreeIndex current_index(sorted_idx_, key_idx_, key_size_, packed_key_, key_schema_, key_num_);

	block_x = (key_num_ < BLOCK_SIZE_X) ? key_num_ : BLOCK_SIZE_X;
	grid_x = (key_num_ - 1)/block_x + 1;
	initialize<<<grid_x, block_x>>>(current_index, table, schema, columns, 0, key_num_ - 1);

	int *tmp_sorted_idx, *tmp;

	checkCudaErrors(hipMalloc(&tmp_sorted_idx, sizeof(int) * key_num_));

	for (int chunk_size = 1; chunk_size <= key_num_/2; chunk_size <<= 1) {
		mergeSort<<<grid_x, block_x>>>(*this, 0, key_num_ - 1, sorted_idx_, tmp_sorted_idx, chunk_size);
		checkCudaErrors(hipDeviceSynchronize());

		tmp = sorted_idx_;
		sorted_idx_ = tmp_sorted_idx;
		tmp_sorted_idx = sorted_idx_;
	}

	checkCudaErrors(hipFree(tmp_sorted_idx));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

extern "C" __global__ void upperBoundSearch(GTreeIndex indexes, GTuple new_tuple, int *key_schema, int key_size, int *entry_idx)
{
	int key_num = indexes.getKeyNum();
	GTreeIndexKey key = indexes.getKeyAtIndex(key_num);

	key.createKey(new_tuple, key_schema, key_size);

	*entry_idx = indexes.upperBound(key, 0, key_num - 1);
}

extern "C" __global__ void lowerBoundSearch(GTreeIndex indexes, GTuple new_tuple, int *key_schema, int key_size, int *entry_idx)
{
	int key_num = indexes.getKeyNum();
	GTreeIndexKey key = indexes.getKeyAtIndex(key_num);

	key.createKey(new_tuple, key_schema, key_size);

	*entry_idx = indexes.lowerBound(key, 0, key_num - 1);
}

void GTreeIndex::addEntry(GTuple new_tuple) {
	int entry_idx, *entry_idx_dev;

	GTreeIndex current_index(sorted_idx_, key_idx_, key_size_, packed_key_, key_schema_, key_num_);

	checkCudaErrors(hipMalloc(&entry_idx_dev, sizeof(int)));
	upperBoundSearch<<<1, 1>>>(current_index, new_tuple, key_idx_, key_size_, entry_idx_dev);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(&entry_idx, entry_idx_dev, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(sorted_idx_ + entry_idx + 1, sorted_idx_ + entry_idx, sizeof(int) * (key_num_ - entry_idx + 1), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(sorted_idx_ + entry_idx, &key_num_, sizeof(int), hipMemcpyHostToDevice));
	key_num_++;
}

/* Add multiple new indexes.
 * New table are already stored in table_ at indexes started from base_idx.
 *
 * */
void GTreeIndex::addBatchEntry(int64_t *table, GColumnInfo *schema, int rows, int columns)
{
	GTreeIndex new_index(sorted_idx_ + key_num_, key_idx_, key_size_, packed_key_ + key_num_ * key_size_, key_schema_, rows);

	new_index.createIndex(table, schema, rows, columns);

	merge(0, key_num_ - 1, key_num_, key_num_ + rows - 1);
	key_num_ += rows;
}

extern "C" __global__ void batchSearchUpper(GTreeIndex indexes, int key_left, int key_right, int left, int right, int *output) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	GTreeIndexKey key;

	for (int i = index; i <= key_right - key_left + 1; i += stride) {
		key = indexes.getKeyAtIndex(i + key_left);

		output[i] = indexes.upperBound(key, left, right);
	}
}

//Search for the lower bounds of an array of keys
extern "C" __global__ void batchSearchLower(GTreeIndex indexes, int key_left, int key_right, int left, int right, int *output) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	GTreeIndexKey key;

	for (int i = index; i <= key_right - key_left + 1; i += stride) {
		key = indexes.getKeyAtIndex(i + key_left);

		output[i] = indexes.lowerBound(key, left, right);
	}
}

extern "C" __global__ void constructWriteLocation(int *location, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i += stride) {
		location[i] += i;
	}
}

extern "C" __global__ void rearrange(int *input, int *output, int *location, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < size; i+= stride) {
		output[location[i]] = input[i];
	}
}

/* Merge new array to the old array
 * Both the new and old arrays are already sorted
 */
void GTreeIndex::merge(int old_left, int old_right, int new_left, int new_right) {
	int old_size, new_size;

	old_size = old_right - old_left + 1;
	new_size = new_right - new_left + 1;

	int block_x, grid_x;

	block_x = (new_size < BLOCK_SIZE_X) ? new_size : BLOCK_SIZE_X;
	grid_x = (new_size - 1) / block_x + 1;

	GTreeIndex current_index(sorted_idx_, key_idx_, key_size_, packed_key_, key_schema_, key_num_);

	int *write_location;

	checkCudaErrors(hipMalloc(&write_location, (old_size + new_size) * sizeof(int)));
	batchSearchUpper<<<grid_x, block_x>>>(current_index, new_left, new_right, old_left, old_right, write_location + old_size);
	constructWriteLocation<<<grid_x, block_x>>>(write_location + old_size, new_size);

	block_x = (old_size < BLOCK_SIZE_X) ? old_size : BLOCK_SIZE_X;
	grid_x = (old_size - 1)/block_x + 1;

	batchSearchLower<<<grid_x, block_x>>>(current_index, old_left, old_right, new_left, new_right, write_location);
	constructWriteLocation<<<grid_x, block_x>>>(write_location, old_size);

	block_x = (old_size + new_size < BLOCK_SIZE_X) ? (old_size + new_size) : BLOCK_SIZE_X;
	grid_x = (old_size + new_size - 1)/block_x + 1;

	int *new_sorted_idx;

	checkCudaErrors(hipMalloc(&new_sorted_idx, (old_size + new_size) * sizeof(int)));
	rearrange<<<grid_x, block_x>>>(sorted_idx_, new_sorted_idx, write_location, old_size + new_size);

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(sorted_idx_));
	sorted_idx_ = new_sorted_idx;
}

void GTreeIndex::removeIndex() {
	checkCudaErrors(hipFree(sorted_idx_));
	checkCudaErrors(hipFree(key_idx_));
}
}

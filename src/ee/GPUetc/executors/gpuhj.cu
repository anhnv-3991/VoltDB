#include "hip/hip_runtime.h"
#include "gpuhj.h"
#include "common/types.h"
#include "GPUetc/storage/gtable.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <string.h>
#include <sys/time.h>
#include <error.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <cudaProfiler.h>
#include <math.h>


#include <inttypes.h>

namespace voltdb {



const uint64_t GPUHJ::MAX_BUCKETS[] = {
	        3,				//0
	        7,				//1
	        13,				//2
	        31,				//3
	        61,				//4
	        127,			//5
	        251,			//6
	        509,			//7
	        1021,			//8
	        2039,			//9
	        4093,			//10
	        8191,			//11
	        16381,			//12
	        32749,			//13
	        65521,			//14
	        131071,			//15
	        262139,			//16
	        524287,			//17
	        1048573,		//18
	        2097143,		//19
	        4194301,		//20
	        8388593,		//21
	        16777213,
	        33554393,
	        67108859,
	        134217689,
	        268435399,
	        536870909,
	        1073741789,
	        2147483647,
	        4294967291,
	        8589934583
	};

GPUHJ::GPUHJ()
{
		outer_table_.block_list = inner_table_.block_list = NULL;
		join_result_ = NULL;
		result_size_ = 0;
		indices_size_ = 0;
		search_exp_size_ = NULL;
		search_exp_num_ = 0;
		indices_ = NULL;
		maxNumberOfBuckets_ = 0;

		search_exp_ = NULL;
		end_expression_.exp = NULL;
		end_expression_.size = 0;
		post_expression_.exp = NULL;
		post_expression_.size = 0;
		initial_expression_.exp = NULL;
		initial_expression_.size = 0;
		skipNullExpr_.exp = NULL;
		skipNullExpr_.size = 0;
		prejoin_expression_.exp = NULL;
		prejoin_expression_.size = 0;
		where_expression_.exp = NULL;
		where_expression_.size = 0;
}

GPUHJ::GPUHJ(GTable outer_table,
				GTable inner_table,
				std::vector<TreeExpression> search_exp,
				std::vector<int> indices,
				TreeExpression end_expression,
				TreeExpression post_expression,
				TreeExpression initial_expression,
				TreeExpression skipNullExpr,
				TreeExpression prejoin_expression,
				TreeExpression where_expression,
				IndexLookupType lookup_type,
				int mSizeIndex)
{
	/**** Table data *********/
	outer_table_ = outer_table;
	inner_table_ = inner_table;
	join_result_ = NULL;
	result_size_ = 0;
	search_exp_num_ = search_exp.size();
	indices_size_ = indices.size();
	lookup_type_ = lookup_type;
	m_sizeIndex_ = mSizeIndex;

	//Fix the size of bucket at 16
	maxNumberOfBuckets_ = MAX_BUCKETS[m_sizeIndex_];

	printf("New M_SIZE_INDEX = %d\n", m_sizeIndex_);

	bool ret = true;
	int tmp_size = 0;

	int *search_exp_size_tmp = (int *)malloc(sizeof(int) * search_exp_num_);
	assert(search_exp_size_tmp != NULL);
	for (int i = 0; i < search_exp_num_; i++) {
		search_exp_size_tmp[i] = search_exp[i].getSize();
		tmp_size += search_exp_size_tmp[i];
	}

	GTreeNode *search_exp_tmp = (GTreeNode *)malloc(sizeof(GTreeNode) * tmp_size);
	assert(search_exp_tmp != NULL);
	GTreeNode *exp_ptr = search_exp_tmp;
	for (int i = 0; i < search_exp_num_; i++) {
		getTreeNodes2(exp_ptr, search_exp[i]);
		exp_ptr += search_exp_size_tmp[i];
	}

	checkCudaErrors(hipMalloc(&search_exp_, tmp_size * sizeof(GTreeNode)));
	checkCudaErrors(hipMalloc(&search_exp_size_, search_exp_num_ * sizeof(int)));
	checkCudaErrors(hipMalloc(&indices_, indices_size_ * sizeof(int)));

	checkCudaErrors(hipMemcpy(search_exp_, search_exp_tmp, tmp_size * sizeof(GTreeNode), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(search_exp_size_, search_exp_size_tmp, search_exp_num_ * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(indices_, &indices[0], indices_size_ * sizeof(int), hipMemcpyHostToDevice));

	free(search_exp_size_tmp);
	free(search_exp_tmp);

	/**** Expression data ****/

	assert(getTreeNodes(&end_expression_, end_expression));

	assert(getTreeNodes(&post_expression_, post_expression));

	assert(getTreeNodes(&initial_expression_, initial_expression));

	assert(getTreeNodes(&skipNullExpr_, skipNullExpr));

	assert(getTreeNodes(&prejoin_expression_, prejoin_expression));

	assert(getTreeNodes(&where_expression_, where_expression));

	int size = 0;

	for (int i = 0; i < indices_size_; i++) {
		switch(inner_table_.schema[indices_[i]].data_type) {
		case VALUE_TYPE_TINYINT: {
			size += sizeof(int8_t);
			break;
		}
		case VALUE_TYPE_SMALLINT: {
			size += sizeof(int16_t);
			break;
		}
		case VALUE_TYPE_INTEGER: {
			size += sizeof(int32_t);
			break;
		}
		case VALUE_TYPE_BIGINT: {
			size += sizeof(int64_t);
			break;
		}
		}
	}

	keySize_ = (size - 1)/8 + 1;
	printf("KEYSIZE = %d\n", keySize_);
}

bool GPUHJ::getTreeNodes(GTree *expression, const TreeExpression tree_expression)
{
	if (tree_expression.getSize() > 0) {
		checkCudaErrors(hipMalloc(&expression->exp, tree_expression.getSize() * sizeof(GTreeNode)));
		checkCudaErrors(hipMemcpy(expression->exp, tree_expression.getNodesArray2(), tree_expression.getSize() * sizeof(GTreeNode), hipMemcpyHostToDevice));
		expression->size = tree_expression.getSize();
	}

	return true;
}

GPUHJ::~GPUHJ()
{
	freeArrays<RESULT>(join_result_);
	freeArrays<GTreeNode>(search_exp_);
	freeArrays<int>(search_exp_size_);
	freeArrays<int>(indices_);
	freeArrays2(end_expression_);
	freeArrays2(post_expression_);
	freeArrays2(initial_expression_);
	freeArrays2(skipNullExpr_);
	freeArrays2(where_expression_);
}

template <typename T> void GPUHJ::freeArrays(T *expression)
{
	if (expression != NULL) {
		free(expression);
	}
}

void GPUHJ::freeArrays2(GTree expression)
{
	if (expression.size > 0) {
		checkCudaErrors(hipFree(expression.exp));
	}
}

void GPUHJ::getResult(RESULT *output) const
{
	memcpy(output, join_result_, sizeof(RESULT) * result_size_);
}

int GPUHJ::getResultSize() const
{
	return result_size_;
}

bool GPUHJ::getTreeNodes2(GTreeNode *expression, const TreeExpression tree_expression)
{
	if (tree_expression.getSize() >= 1)
		tree_expression.getNodesArray(expression);

	return true;
}

void GPUHJ::debug(void)
{
//	std::cout << "Size of outer table = " << outer_rows_ << std::endl;
//	if (outer_rows_ != 0) {
//		std::cout << "Outer table" << std::endl;
//		for (int i = 0; i < outer_rows_; i++) {
//			for (int j = 0; j < MAX_GNVALUE; j++) {
//				NValue tmp;
//				setNValue(&tmp, outer_table_[i * outer_cols_ + j]);
//				std::cout << tmp.debug().c_str() << std::endl;
//			}
//		}
//	} else
//		std::cout << "Empty outer table" << std::endl;
//
//	std::cout << "Size of inner table =" << inner_rows_ << std::endl;
//	if (inner_rows_ != 0) {
//		for (int i = 0; i < inner_rows_; i++) {
//			for (int j = 0; j < MAX_GNVALUE; j++) {
//				NValue tmp;
//				setNValue(&tmp, inner_table_[i * inner_cols_ + j]);
//				std::cout << tmp.debug().c_str() << std::endl;
//			}
//		}
//	} else
//		std::cout << "Empty inner table" << std::endl;
//
//	std::cout << "Size of end_expression = " << end_size_ << std::endl;
//	if (end_size_ != 0) {
//		std::cout << "Content of end_expression" << std::endl;
//		debugGTrees(end_expression_, end_size_);
//	} else
//		std::cout << "Empty end expression" << std::endl;
//
//	std::cout << "Size of post_expression = " << post_size_ << std::endl;
//	if (post_size_ != 0) {
//		std::cout << "Content of post_expression" << std::endl;
//		debugGTrees(post_expression_, post_size_);
//	} else
//		std::cout << "Empty post expression" << std::endl;
//
//	std::cout << "Size of initial_expression = " << initial_size_ << std::endl;
//	if (initial_size_ != 0) {
//		std::cout << "Content of initial_expression" << std::endl;
//		debugGTrees(initial_expression_, initial_size_);
//	} else
//		std::cout << "Empty initial expression" << std::endl;
//
//	std::cout << "Size of skip null expression = " << skipNull_size_ << std::endl;
//	if (skipNull_size_ != 0) {
//		std::cout << "Content of skip null_expression" << std::endl;
//		debugGTrees(skipNullExpr_, skipNull_size_);
//	} else
//		std::cout << "Empty skip null expression" << std::endl;
//
//	std::cout << "Size of prejoin_expression = " << prejoin_size_ << std::endl;
//	if (prejoin_size_ != 0) {
//		std::cout << "Content of prejoin_expression" << std::endl;
//		debugGTrees(prejoin_expression_, prejoin_size_);
//	} else
//		std::cout << "Empty prejoin expression " << std::endl;
//
//	std::cout << "Size of where expression = " << where_size_ << std::endl;
//	if (where_size_ != 0) {
//		std::cout << "Content of where_expression" << std::endl;
//		debugGTrees(where_expression_, where_size_);
//	} else
//		std::cout << "Empty where expression" << std::endl;
//
//	std::cout << "Size of search_exp_ array = " << search_exp_num_ << std::endl;
//	int search_exp_ptr = 0;
//	if (search_exp_num_ != 0) {
//		std::cout << "Content of search_exp" << std::endl;
//		for (int i = 0; i < search_exp_num_; i++) {
//			std::cout << "search_exp[" << i << std::endl;
//			debugGTrees(search_exp_ + search_exp_ptr, search_exp_size_[i]);
//			search_exp_ptr += search_exp_size_[i];
//		}
//	} else
//		std::cout << "Empty search keys array" << std::endl;
//
//	std::cout << "Size of innner_indices = " << indices_size_ << std::endl;
//	if (indices_size_ != 0) {
//		std::cout << "Content of inner indices" << std::endl;
//		for (int i = 0; i < indices_size_; i++) {
//			std::cout << "indices[" << i << "] = " << indices_[i] << std::endl;
//		}
//	} else
//		std::cout << "Empty indices array" << std::endl;
}




uint GPUHJ::getPartitionSize() const
{
//	return PART_SIZE_;
	uint part_size = DEFAULT_PART_SIZE_;
//	uint outer_size = outer_rows_;
//	uint inner_size = inner_rows_;
//	uint bigger_tuple_size = (outer_size > inner_size) ? outer_size : inner_size;
//
//	if (bigger_tuple_size < part_size) {
//		return bigger_tuple_size;
//	}
//
//	for (uint i = 32768; i <= DEFAULT_PART_SIZE_; i = i * 2) {
//		if (bigger_tuple_size < i) {
//			part_size = i;
//			break;
//		}
//	}
//
//	printf("getPartitionSize: PART SIZE = %d\n", part_size);
	return part_size;
}


bool GPUHJ::join()
{
	return false;
//
//	checkCudaErrors(hipProfilerStart());
//	GTable outer_chunk, inner_chunk;
//	ulong *index_count, jr_size;
//	RESULT *jresult_dev;
//	int block_x, grid_x, grid_y;
//	struct timeval start_all, end_all;
//#if defined(FUNC_CALL_) && defined(POST_EXP_)
//	GNValue *stack;
//#elif defined(POST_EXP_)
//	int64_t *val_stack;
//	ValueType *type_stack;
//#endif
//	int partition_size, size_of_buckets, bucketStride;
//	double tmp; //For calculating block size and grid size in power of 2
//	struct timeval inner_pack_start, inner_pack_end, inner_hash_count_start, inner_hash_count_end, inner_hash_start, inner_hash_end, inner_prefix_start, inner_prefix_end;
//	std::vector<unsigned long> inner_pack, inner_hasher;
//
//	struct timeval outer_pack_start, outer_pack_end, outer_hash_count_start, outer_hash_count_end, outer_prefix_start, outer_prefix_end, outer_hash_start, outer_hash_end;
//	std::vector<unsigned long> outer_pack, outer_hasher;
//
//	struct timeval index_count_start, index_count_end, prefix_start, prefix_end, join_start, join_end, rebalance_start, rebalance_end, remove_start, remove_end;
//	std::vector<unsigned long> index_hcount, prefix_sum, join_time, rebalance_cost, remove_empty;
//
//	gettimeofday(&start_all, NULL);
//
//	/******* Hash the outer table *******/
//#ifdef METHOD_1_
//	partition_size = getPartitionSize();
//	GHashNode *inner_hash_host;
//	GHashNode outer_hash_dev, inner_hash_dev;
//	bool *inner_hashed;
//	int part_num_inner;
//
//	part_num_inner = (inner_rows_ - 1)/partition_size + 1;
////	tmp = part_num_inner;
////	m_sizeIndex_ -= (int)log2(tmp);
////
////	maxNumberOfBuckets_ = MAX_BUCKETS[m_sizeIndex_];
//	printf("New m_sizeIndex = %d\n", m_sizeIndex_);
//	inner_hash_host = (GHashNode *)malloc(sizeof(GHashNode) * part_num_inner);
//	inner_hashed = (bool*)malloc(sizeof(bool) * part_num_inner);
//
//
//	outer_hash_dev.bucket_num = maxNumberOfBuckets_;
//	outer_hash_dev.key_size = keySize_;
//
//
//	for (int i = 0; i < part_num_inner; i++) {
//		inner_hashed[i] = false;
//		inner_hash_host[i].bucket_num = maxNumberOfBuckets_;
//		inner_hash_host[i].key_size = keySize_;
//		inner_hash_host[i].size = partition_size;
//		inner_hash_host[i].bucket_location = (int*)malloc(sizeof(int) * (maxNumberOfBuckets_ + 1));
//		inner_hash_host[i].hashed_idx = (int*)malloc(sizeof(int) * partition_size);
//		inner_hash_host[i].hashed_key = (uint64_t*)malloc(sizeof(uint64_t) * partition_size * keySize_);
//	}
//
//	inner_hash_dev.bucket_num = maxNumberOfBuckets_;
//	inner_hash_dev.key_size = keySize_;
//
//	checkCudaErrors(hipMalloc(&outer_key, sizeof(uint64_t) * partition_size * keySize_));
//	checkCudaErrors(hipMalloc(&(outer_hash_dev.hashed_idx), sizeof(int) * partition_size));
//	checkCudaErrors(hipMalloc(&(outer_hash_dev.hashed_key), sizeof(uint64_t) * partition_size * keySize_));
//	checkCudaErrors(hipMalloc(&(outer_hash_dev.bucket_location), sizeof(int) * (maxNumberOfBuckets_ + 1)));
//
//	checkCudaErrors(hipMalloc(&inner_key, sizeof(uint64_t) * partition_size * keySize_));
//	checkCudaErrors(hipMalloc(&(inner_hash_dev.hashed_idx), sizeof(int) * partition_size));
//	checkCudaErrors(hipMalloc(&(inner_hash_dev.hashed_key), sizeof(uint64_t) * partition_size * keySize_));
//	checkCudaErrors(hipMalloc(&(inner_hash_dev.bucket_location), sizeof(int) * (maxNumberOfBuckets_ + 1)));
//
//	tmp = (outer_rows_ - 1)/maxNumberOfBuckets_ + 1;
//	size_of_buckets = (int)pow(2, (int)(log2(tmp)));
//
//	checkCudaErrors(hipMalloc(&index_count, sizeof(ulong) * (partition_size + 1)));
//	checkCudaErrors(hipMalloc(&outer_dev, sizeof(GNValue) * partition_size * outer_cols_));
//	checkCudaErrors(hipMalloc(&inner_dev, sizeof(GNValue) * partition_size * inner_cols_));
//
//	ResBound *in_bound;
//
//	checkCudaErrors(hipMalloc(&in_bound, sizeof(ResBound) * partition_size));
//
//	printf("Start Joining\n");
//	for (int base_outer_idx = 0; base_outer_idx < outer_rows_; base_outer_idx += partition_size) {
//		/*** Hash the outer partition ***/
//		int outer_part_size = (base_outer_idx + partition_size < outer_rows_) ? partition_size : (outer_rows_ - base_outer_idx);
//
//		checkCudaErrors(hipMemcpy(outer_dev, outer_table_ + base_outer_idx * outer_cols_, sizeof(GNValue) * outer_part_size * outer_cols_, hipMemcpyHostToDevice));
//
//		gettimeofday(&outer_pack_start, NULL);
//		PackSearchKeyWrapper(outer_dev, outer_part_size, outer_cols_, outer_key, search_exp_dev, search_exp_size, search_exp_num_, keySize_);
//		gettimeofday(&outer_pack_end, NULL);
//		outer_pack.push_back(timeDiff(outer_pack_start, outer_pack_end));
//
//		gettimeofday(&outer_hash_start, NULL);
//		outer_hash_dev.size = outer_part_size;
//		GhashWrapper(outer_key, outer_hash_dev);
//		gettimeofday(&outer_hash_end, NULL);
//		outer_hasher.push_back(timeDiff(outer_hash_start, outer_hash_end));
//
//		for (int base_inner_idx = 0, j = 0; base_inner_idx < inner_rows_; base_inner_idx += partition_size, j++) {
//			int inner_part_size = (base_inner_idx + partition_size < inner_rows_) ? partition_size : (inner_rows_ - base_inner_idx);
//
//			checkCudaErrors(hipMemcpy(inner_dev, inner_table_ + base_inner_idx * inner_cols_, sizeof(GNValue) * inner_part_size * inner_cols_, hipMemcpyHostToDevice));
//
//			if (!inner_hashed[j]) {
//				inner_hashed[j] = true;
//				gettimeofday(&inner_pack_start, NULL);
//				PackKeyWrapper(inner_dev, inner_part_size, inner_cols_, indices_dev, indices_size_, inner_key, keySize_);
//				gettimeofday(&inner_pack_end, NULL);
//				inner_pack.push_back(timeDiff(inner_pack_start, inner_pack_end));
//
//				gettimeofday(&inner_hash_start, NULL);
//				inner_hash_dev.size = inner_part_size;
//				GhashWrapper(inner_key, inner_hash_dev);
//				gettimeofday(&inner_hash_end, NULL);
//				inner_hasher.push_back(timeDiff(inner_hash_start, inner_hash_end));
//
//				checkCudaErrors(hipMemcpy(inner_hash_host[j].bucket_location, inner_hash_dev.bucket_location, sizeof(int) * (maxNumberOfBuckets_ + 1), hipMemcpyDeviceToHost));
//				checkCudaErrors(hipMemcpy(inner_hash_host[j].hashed_idx, inner_hash_dev.hashed_idx, sizeof(int) * inner_part_size, hipMemcpyDeviceToHost));
//				checkCudaErrors(hipMemcpy(inner_hash_host[j].hashed_key, inner_hash_dev.hashed_key, sizeof(uint64_t) * inner_part_size * keySize_, hipMemcpyDeviceToHost));
//				inner_hash_host[j].size = inner_part_size;
//			} else {
//				checkCudaErrors(hipMemcpy(inner_hash_dev.bucket_location, inner_hash_host[j].bucket_location, sizeof(int) * (maxNumberOfBuckets_ + 1), hipMemcpyHostToDevice));
//				checkCudaErrors(hipMemcpy(inner_hash_dev.hashed_idx, inner_hash_host[j].hashed_idx, sizeof(int) * inner_part_size, hipMemcpyHostToDevice));
//				checkCudaErrors(hipMemcpy(inner_hash_dev.hashed_key, inner_hash_host[j].hashed_key, sizeof(uint64_t) * inner_part_size * keySize_, hipMemcpyHostToDevice));
//				inner_hash_dev.size = inner_hash_host[j].size;
//			}
//
//			gettimeofday(&index_count_start, NULL);
//#if !defined(DECOMPOSED1_) && !defined(DECOMPOSED2_)
//			IndexCountWrapper(outer_hash_dev, inner_hash_dev, index_count, partition_size + 1);
//#else
//			IndexCountWrapper2(outer_hash_dev, inner_hash_dev, index_count, in_bound);
//#endif
//			gettimeofday(&index_count_end, NULL);
//			index_hcount.push_back(timeDiff(index_count_start, index_count_end));
//
//#if !defined(DECOMPOSED1_) && !defined(DECOMPOSED2_)
//			gettimeofday(&prefix_start, NULL);
//			ExclusiveScanWrapper(index_count, partition_size + 1, &jr_size);
//			gettimeofday(&prefix_end, NULL);
//			prefix_sum.push_back(timeDiff(prefix_start, prefix_end));
//
//			if (jr_size < 0) {
//				printf("Scanning failed\n");
//				return false;
//			}
//
//			if (jr_size == 0) {
//				continue;
//			}
//
//			checkCudaErrors(hipMalloc(&jresult_dev, jr_size * sizeof(RESULT)));
//
//			gettimeofday(&join_start, NULL);
//			HashJoinWrapper(outer_dev, inner_dev,
//								outer_cols_, inner_cols_,
//								end_dev, end_size_,
//								post_dev, post_size_,
//								outer_hash_dev, inner_hash_dev,
//								base_outer_idx, base_inner_idx,
//								index_count, outer_part_size,
//								jresult_dev);
//			gettimeofday(&join_end, NULL);
//			join_time.push_back(timeDiff(join_start, join_end));
//
//			join_result_ = (RESULT *)realloc(join_result_, (result_size_ + jr_size) * sizeof(RESULT));
//
//			checkCudaErrors(hipMemcpy(join_result_ + result_size_, jresult_dev, jr_size * sizeof(RESULT), hipMemcpyDeviceToHost));
//			checkCudaErrors(hipFree(jresult_dev));
//			result_size_ += jr_size;
//			jr_size = 0;
//#else
//			RESULT *tmp_bound, *out_bound;
//			ulong out_size;
//			ulong *exp_psum;
//
//			gettimeofday(&rebalance_start, NULL);
//#ifdef DECOMPOSED1_
//			HRebalance2(index_count, in_bound, inner_hash_dev, &tmp_bound, outer_part_size + 1, &out_size);
//#else
//			HRebalance(index_count, in_bound, inner_hash_dev, &tmp_bound, outer_part_size, &out_size);
//#endif
//			gettimeofday(&rebalance_end, NULL);
//			rebalance_cost.push_back(timeDiff(rebalance_start, rebalance_end));
//
//			if (out_size == 0) {
//				continue;
//			}
//
//			checkCudaErrors(hipMalloc(&exp_psum, (out_size + 1) * sizeof(ulong)));
//			checkCudaErrors(hipMalloc(&out_bound, out_size * sizeof(RESULT)));
//
//			gettimeofday(&join_start, NULL);
//
//#ifndef SHARED_
//			ExpressionFilterWrapper2(outer_dev, inner_dev,
//										tmp_bound, out_bound,
//										exp_psum, out_size,
//										outer_cols_, inner_cols_,
//										end_dev, end_size_,
//										post_dev, post_size_,
//										where_dev, where_size_,
//										base_outer_idx, base_inner_idx);
//#else
//			ExpressionFilterWrapper3(outer_dev, inner_dev,
//										tmp_bound, out_bound,
//										exp_psum, out_size,
//										outer_cols_, inner_cols_,
//										end_dev, end_size_,
//										post_dev, post_size_,
//										where_dev, where_size_,
//										base_outer_idx, base_inner_idx);
//#endif
//			gettimeofday(&join_end, NULL);
//			join_time.push_back(timeDiff(join_start, join_end));
//
//			gettimeofday(&prefix_start, NULL);
//			ExclusiveScanWrapper(exp_psum, out_size + 1, &jr_size);
//			gettimeofday(&prefix_end, NULL);
//
//			prefix_sum.push_back(timeDiff(prefix_start, prefix_end));
//
//			checkCudaErrors(hipFree(tmp_bound));
//
//			if (jr_size == 0) {
//				printf("EMPTY RESULT******************************************\n");
//				checkCudaErrors(hipFree(exp_psum));
//				checkCudaErrors(hipFree(out_bound));
//				continue;
//			}
//
//			checkCudaErrors(hipMalloc(&jresult_dev, jr_size * sizeof(RESULT)));
//
//			gettimeofday(&remove_start, NULL);
//			RemoveEmptyResultWrapper2(jresult_dev, out_bound, exp_psum, out_size);
//			gettimeofday(&remove_end, NULL);
//			remove_empty.push_back(timeDiff(remove_start, remove_end));
//			join_result_ = (RESULT *)realloc(join_result_, (result_size_ + jr_size) * sizeof(RESULT));
//
//			checkCudaErrors(hipMemcpy(join_result_ + result_size_, jresult_dev, jr_size * sizeof(RESULT), hipMemcpyDeviceToHost));
//
//			checkCudaErrors(hipFree(exp_psum));
//			checkCudaErrors(hipFree(out_bound));
//			checkCudaErrors(hipFree(jresult_dev));
//			result_size_ += jr_size;
//			jr_size = 0;
//#endif
//		}
//	}
//
//#else
//	partition_size = getPartitionSize();
//	checkCudaErrors(hipMalloc(&index_count, sizeof(ulong) * (partition_size + 1)));
//
//	ResBound *in_bound;
//
//	checkCudaErrors(hipMalloc(&in_bound, sizeof(ResBound) * partition_size));
//
//	printf("Start Joining\n");
//
//	outer_chunk.column_num = outer_table_.column_num;
//	outer_chunk.schema = outer_table_.schema;
//	inner_chunk.column_num = inner_table_.column_num;
//	inner_chunk.schema = inner_table_.schema;
//	int64_t *outer_key = NULL;
//	for (int outer_idx = 0; outer_idx < outer_table_.block_num; outer_idx++) {
//		/*** Hash the outer partition ***/
//		outer_chunk.block_list = outer_table_.block_list + outer_idx;
//
//		for (int inner_idx = 0; inner_idx < inner_table_.block_num; inner_idx++) {
//
//			inner_chunk.block_list = inner_table_.block_list + inner_idx;
//
//			gettimeofday(&index_count_start, NULL);
//			//IndexCountLegacyWrapper(outer_key, outer_chunk.block_list->rows, inner_hash_dev, index_count, in_bound);
//			gettimeofday(&index_count_end, NULL);
//
//			index_hcount.push_back(timeDiff(index_count_start, index_count_end));
//
//#if !defined(DECOMPOSED1_) && !defined(DECOMPOSED2_)
//			gettimeofday(&prefix_start, NULL);
//			ExclusiveScanWrapper(index_count, partition_size + 1, &jr_size);
//			gettimeofday(&prefix_end, NULL);
//
//			prefix_sum.push_back(timeDiff(prefix_start, prefix_end));
//
//			if (jr_size < 0) {
//				printf("Scanning failed\n");
//				return false;
//			}
//
//			if (jr_size == 0) {
//				continue;
//			}
//
//			checkCudaErrors(hipMalloc(&jresult_dev, jr_size * sizeof(RESULT)));
//
//			gettimeofday(&join_start, NULL);
//			HashJoinLegacyWrapper(outer_dev, inner_dev,
//									outer_cols_, inner_cols_,
//									outer_part_size, outer_key,
//									end_dev, end_size_,
//									post_dev, post_size_, inner_hash_dev,
//									base_outer_idx, base_inner_idx,
//									index_count, in_bound,
//									jresult_dev);
//			gettimeofday(&join_end, NULL);
//			join_time.push_back(timeDiff(join_start, join_end));
//#else
//			RESULT *tmp_bound, *out_bound;
//			ulong out_size;
//			ulong *exp_psum;
//
//			gettimeofday(&rebalance_start, NULL);
//#ifdef DECOMPOSED1_
//			//HRebalance(index_count, in_bound, inner_hash_dev, &tmp_bound, outer_part_size, &out_size);
//#else
//			HRebalance2(index_count, in_bound, inner_hash_dev, &tmp_bound, outer_part_size + 1, &out_size);
//#endif
//			gettimeofday(&rebalance_end, NULL);
//			rebalance_cost.push_back(timeDiff(rebalance_start, rebalance_end));
//
//			if (out_size == 0) {
//				continue;
//			}
//
//			printf("out_size = %lu\n", out_size);
//			checkCudaErrors(hipMalloc(&exp_psum, (out_size + 1) * sizeof(ulong)));
//			checkCudaErrors(hipMalloc(&out_bound, out_size * sizeof(RESULT)));
//
//			gettimeofday(&join_start, NULL);
////			ExpressionFilterWrapper2(outer_dev, inner_dev,
////										tmp_bound, out_bound,
////										exp_psum, out_size,
////										outer_cols_, inner_cols_,
////										end_dev, end_size_,
////										post_dev, post_size_,
////										where_dev, where_size_,
////										base_outer_idx, base_inner_idx);
//			gettimeofday(&join_end, NULL);
//			join_time.push_back(timeDiff(join_start, join_end));
//
//			gettimeofday(&prefix_start, NULL);
//			ExclusiveScanWrapper(exp_psum, out_size + 1, &jr_size);
//			gettimeofday(&prefix_end, NULL);
//
//			prefix_sum.push_back(timeDiff(prefix_start, prefix_end));
//
//			checkCudaErrors(hipFree(tmp_bound));
//
//			if (jr_size == 0) {
//				checkCudaErrors(hipFree(exp_psum));
//				checkCudaErrors(hipFree(out_bound));
//				continue;
//			}
//
//			checkCudaErrors(hipMalloc(&jresult_dev, jr_size * sizeof(RESULT)));
//
//			gettimeofday(&remove_start, NULL);
//			RemoveEmptyResultWrapper2(jresult_dev, out_bound, exp_psum, out_size);
//			gettimeofday(&remove_end, NULL);
//			remove_empty.push_back(timeDiff(remove_start, remove_end));
//#endif
//
//			join_result_ = (RESULT *)realloc(join_result_, (result_size_ + jr_size) * sizeof(RESULT));
//
//			checkCudaErrors(hipMemcpy(join_result_ + result_size_, jresult_dev, jr_size * sizeof(RESULT), hipMemcpyDeviceToHost));
//#ifdef DECOMPOSED1_
//			checkCudaErrors(hipFree(exp_psum));
//			checkCudaErrors(hipFree(out_bound));
//#endif
//			checkCudaErrors(hipFree(jresult_dev));
//			result_size_ += jr_size;
//			jr_size = 0;
//		}
//	}
//
//#endif
//
//	gettimeofday(&end_all, NULL);
//	unsigned long inner_pack_final, inner_hash_final;
//	unsigned long outer_pack_final, outer_hash_final;
//	unsigned long index_count_final, prefix_sum_final, join_final;
//	unsigned long rebalance_final, remove_empty_total;
//
//	inner_pack_final = 0;
//	for (int i = 0; i < inner_pack.size(); i++) {
//		printf("InnerPack time at %d is %lu\n", i, inner_pack[i]);
//		inner_pack_final += inner_pack[i];
//	}
//
//
//	inner_hash_final = 0;
//	for (int i = 0; i < inner_hasher.size(); i++) {
//		inner_hash_final += inner_hasher[i];
//	}
//
//	outer_pack_final = 0;
//	for (int i = 0; i < outer_pack.size(); i++) {
//		outer_pack_final += outer_pack[i];
//	}
//
//	outer_hash_final = 0;
//	for (int i = 0; i < outer_hasher.size(); i++) {
//		outer_hash_final += outer_hasher[i];
//	}
//
//	index_count_final = 0;
//	for (int i = 0; i < index_hcount.size(); i++) {
//		//printf("index count time = %lu\n", index_hcount[i]);
//		index_count_final += index_hcount[i];
//	}
//
//	prefix_sum_final = 0;
//	for (int i = 0; i < prefix_sum.size(); i++) {
//		//printf("Prefix sum time = %lu\n", prefix_sum[i]);
//		prefix_sum_final += prefix_sum[i];
//	}
//
//	rebalance_final = 0;
//	for (int i = 0; i < rebalance_cost.size(); i++) {
//		rebalance_final += rebalance_cost[i];
//	}
//
//	remove_empty_total = 0;
//	for (int i = 0; i < remove_empty.size(); i++) {
//		remove_empty_total += remove_empty[i];
//	}
//
//	join_final = 0;
//	for (int i = 0; i < join_time.size(); i++) {
//		join_final += join_time[i];
//	}
//
//	ulong inner_hash_total, outer_hash_total, join_total, data_copy, total;
//
//	inner_hash_total = inner_pack_final + inner_hash_final;
//	outer_hash_total = outer_pack_final + outer_hash_final;
//	join_total = index_count_final + prefix_sum_final + join_final + remove_empty_total;
//	total = timeDiff(start_all, end_all);
//	data_copy = total - inner_hash_total - outer_hash_total - join_total;
//
//	printf("\n*** Execution time *****************************\n"
//			"Inner Pack: %lu\n"
//			"Inner Hash: %lu\n\n"
//
//			"Outer Pack: %lu\n"
//			"Outer Hash: %lu\n\n"
//
//			"index Count: %lu\n"
//			"prefix_sum: %lu\n"
//			"Join: %lu\n"
//			"*************************************************\n"
//			"Inner hash Total: %lu\n"
//			"Outer hash Total: %lu\n"
//#ifdef DECOMPOSED1_
//			"Rebalance total: %lu\n"
//#endif
//			"Exp evaluation: %lu\n"
//			"Remove empty total: %lu\n"
//			"Data copy: %lu\n"
//			"Total time including data copy: %lu\n"
//			"Total time excluding data copy: %lu\n"
//			"Total time: %lu\n", inner_pack_final, inner_hash_final,
//								outer_pack_final, outer_hash_final,
//								index_count_final, prefix_sum_final, join_final,
//								inner_hash_total, outer_hash_total,
//#ifdef DECOMPOSED1_
//								rebalance_final,
//#endif
//								join_total, remove_empty_total, data_copy, total - inner_hash_total, total - inner_hash_total - data_copy, total);
//
//
//	checkCudaErrors(hipFree(outer_dev));
//	checkCudaErrors(hipFree(inner_dev));
//
//	if (initial_size_ > 0)
//		checkCudaErrors(hipFree(initial_dev));
//
//	if (end_size_ > 0)
//		checkCudaErrors(hipFree(end_dev));
//
//	if (post_size_ > 0)
//		checkCudaErrors(hipFree(post_dev));
//
//	if (where_size_ > 0)
//		checkCudaErrors(hipFree(where_dev));
//
//	checkCudaErrors(hipFree(search_exp_dev));
//	checkCudaErrors(hipFree(search_exp_size));
//	checkCudaErrors(hipFree(indices_dev));
//	checkCudaErrors(hipFree(index_count));
//
//#ifdef METHOD_1_
//	checkCudaErrors(hipFree(outer_key));
//	checkCudaErrors(hipFree(inner_key));
//	checkCudaErrors(hipFree(outer_hash_dev.bucket_location));
//	checkCudaErrors(hipFree(outer_hash_dev.hashed_idx));
//	checkCudaErrors(hipFree(outer_hash_dev.hashed_key));
//	checkCudaErrors(hipFree(inner_hash_dev.bucket_location));
//	checkCudaErrors(hipFree(inner_hash_dev.hashed_idx));
//	checkCudaErrors(hipFree(inner_hash_dev.hashed_key));
//
//#ifdef DECOMPOSED1_
//	checkCudaErrors(hipFree(in_bound));
//#endif
//
//	int max = 0, maxId = 0, blockMax;
//	for (int i = 0; i < part_num_inner; i++) {
//		for (int k = 0; k < maxNumberOfBuckets_; k++) {
//			if (inner_hash_host[i].bucket_location[k + 1] - inner_hash_host[i].bucket_location[k] > max) {
//				max = inner_hash_host[i].bucket_location[k + 1] - inner_hash_host[i].bucket_location[k];
//				maxId = k;
//				blockMax = i;
//			}
//		}
//		free(inner_hash_host[i].bucket_location);
//		free(inner_hash_host[i].hashed_idx);
//		free(inner_hash_host[i].hashed_key);
//	}
//
//	printf("Max bucket of Outer at index %d and block %d is %d\n", maxId, blockMax, max);
//
//	free(inner_hash_host);
//	free(inner_hashed);
//#else
//	checkCudaErrors(hipFree(outer_key));
//	checkCudaErrors(hipFree(inner_key));
//	checkCudaErrors(hipFree(inner_hash_dev.bucket_location));
//	checkCudaErrors(hipFree(inner_hash_dev.hashed_idx));
//	checkCudaErrors(hipFree(inner_hash_dev.hashed_key));
//
//
//	for (int i = 0; i < part_num_inner; i++) {
//		free(inner_hash_host[i].bucket_location);
//		free(inner_hash_host[i].hashed_idx);
//		free(inner_hash_host[i].hashed_key);
//	}
//	free(inner_hash_host);
//	free(inner_hashed);
//
//#endif
//
//	checkCudaErrors(hipProfilerStop());
//
//	return true;
}

extern "C" __global__ void EvaluateSearchPredicate(GTable outer_table, GTreeNode *search_keys, int *search_size, int search_num,
													int64_t *val_stack, ValueType *type_stack, GTable output, GHashIndex output_index)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < rows; i += stride) {
		GTuple tuple_res(output, i);
		GTuple outer_tuple(outer_table, i);

		for (int j = 0, search_ptr = 0; j < search_num; search_ptr += search_size[j], j++) {
			GExpression search_exp(search_keys + search_ptr, search_size[j]);
			GNValue eval_result = search_exp.evaluate(&outer_tuple, NULL, val_stack, type_stack, stride);

			tuple_res.attachColumn(eval_result);
		}

		output_index.insertKeyTupleNoSort(tuple_res, i);
	}
}

extern "C" __global__ void indexCount(GHashIndex outer_index, GHashIndex inner_index, ulong *index_count, ResBound *out_bound)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int outer_rows = outer_index.getKeyRows();

	for (int i = index; i < outer_rows; i += stride) {
		GHashIndexKey key(outer_index, i);
		int bucket_id = key.KeyHasher();

		out_bound[i].left = inner_index.getBucketLocation(bucket_id);
		out_bound[i].right = inner_index.getBucketLocation(bucket_id + 1);

		index_count[i] = out_bound[i].right - out_bound[i].left + 1;
	}
}

void GPUHJ::IndexCount(ulong *index_count, ResBound *out_bound)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1)/block_x + 1;

	GColumnInfo *search_schema;

	checkCudaErrors(hipMalloc(&search_schema, sizeof(GColumnInfo) * search_size_num));
	GTable search_table(outer_table_.getDatabaseId(), NULL, search_schema, search_size_num, outer_table_.getCurrentRowNum());
	GHashIndex tmp_index(outer_table_.getCurrentRowNum(), search_size_num);

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * outer_rows * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * outer_rows * MAX_STACK_SIZE));
	EvaluateSearchPredicate<<<grid_x, block_x>>>(outer_table_, search_exp_, search_exp_size_, search_exp_num_, val_stack, type_stack, search_table, tmp_index);
	GHashIndex inner_index = (GHashIndex)(inner_table_.getIndex());
	indexCount<<<grid_x, block_x>>>(tmp_index, inner_index, index_count, out_bound);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(search_schema));
	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

void GPUHJ::IndexCount(ulong *index_count, ResBound *out_bound, hipStream_t stream)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1)/block_x + 1;

	GColumnInfo *search_schema;

	checkCudaErrors(hipMalloc(&search_schema, sizeof(GColumnInfo) * search_size_num));
	GTable search_table(outer_table_.getDatabaseId(), NULL, search_schema, search_size_num, outer_table_.getCurrentRowNum());

	GHashIndex tmp_index(outer_table_.getCurrentRowNum(), search_size_num);

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * outer_rows * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * outer_rows * MAX_STACK_SIZE));

	EvaluateSearchPredicate<<<grid_x, block_x, 0, stream>>>(outer_table_, search_exp_, search_exp_size_, search_exp_num_, val_stack, type_stack, search_table, tmp_index);

	GHashIndex inner_index = (GHashIndex)(inner_table_.getIndex());
	indexCount<<<grid_x, block_x, 0, stream>>>(tmp_index, inner_index, index_count, out_bound);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(search_schema));
	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

extern "C" __global__ void hashJoinLegacy(GTable outer, GTable inner,
											RESULT *in_bound, RESULT *out_bound,
											ulong *mark_location, int size,
											GExpression end_exp, GExpression post_exp, GExpression where_exp,
											int64_t *val_stack, ValueType *type_stack
											)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	GNValue res;

	for (int i = index; i < size; i += offset) {
		GTuple outer_tuple(outer, in_bound[i].lkey);
		GTuple inner_tuple(inner, in_bound[i].rkey);
		res = GNValue::getTrue();

		res = (end_exp.getSize() > 0) ? end_exp.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;
		res = (post_exp.getSize() > 0 && res.isTrue()) ? post_exp.evaluate(&outer_tuple, inner_tuple, val_stack + index, type_stack + index, offset) : res;
		res = (where_exp.getSize() > 0 && res.isTrue()) ? where_exp.evaluate(&outer_tuple, inner_tuple, val_stack + index, type_stack + index, offset) : res;

		out_bound[i].lkey = (res.isTrue()) ? in_bound[i].lkey : (-1);
		out_bound[i].rkey = (res.isTrue()) ? in_bound[i].rkey : (-1);
		mark_location[i] = (res.isTrue()) ? 1 : 0;
	}

	if (index == 0) {
		mark_location[size] = 0;
	}
}

void GPUHJ::HashJoinLegacy(RESULT *in_bound, RESULT *out_bound, ulong *mark_location, int size)
{
	int partition_size = DEFAULT_PART_SIZE_;
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size < partition_size) ? (size - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;


	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	HashJoinLegacy<<<grid_size, block_size>>>(outer_table_, inner_table_,
												in_bound, out_bound,
												mark_location, size,
												end_expression_, post_expression_, where_exp_,
												val_stack,
												type_stack);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

void GPUHJ::HashJoinLegacy(RESULT *in_bound, RESULT *out_bound, ulong *mark_location, int size, hipStream_t stream)
{
	int partition_size = DEFAULT_PART_SIZE_;
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size < partition_size) ? (size - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;


	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	HashJoinLegacy<<<grid_size, block_size, 0, stream>>>(outer_table_, inner_table_,
												in_bound, out_bound,
												mark_location, size,
												end_expression_, post_expression_, where_exp_,
												val_stack,
												type_stack);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}


__global__ void HDecompose(RESULT *output, ResBound *in_bound, int *sorted_idx, ulong *in_location, ulong *local_offset, int size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = index; i < size; i += blockDim.x * gridDim.x) {
		output[i].lkey = in_bound[in_location[i]].outer;
		output[i].rkey = in_hash.hashed_idx[in_bound[in_location[i]].left + local_offset[i]];
	}
}

void GPUHJ::decompose(RESULT *output, ResBound *in_bound, ulong *in_location, ulong *local_offset, int size)
{
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	int *sorted_idx = inner_table_.getIndex().getSortedIdx();

	HDecompose<<<grid_size, block_size>>>(output, in_bound, sorted_idx, in_location, local_offset, size);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void GPUHJ::decompose(RESULT *output, ResBound *in_bound, ulong *in_location, ulong *local_offset, int size, hipStream_t stream)
{
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	int *sorted_idx = inner_table_.getIndex().getSortedIdx();

	HDecompose<<<grid_size, block_size, 0, stream>>>(output, in_bound, sorted_idx, in_location, local_offset, size);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipStreamSynchronize(stream));
}

void GPUHJ::Rebalance(ulong *index_count, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size)
{
	ExclusiveScanWrapper(index_count, in_size, out_size);

	if (*out_size == 0) {
		return;
	}

	ulong *location;

	checkCudaErrors(hipMalloc(&location, sizeof(ulong) * (*out_size)));
	checkCudaErrors(hipMemset(location, 0, sizeof(ulong) * (*out_size)));
	checkCudaErrors(hipDeviceSynchronize());


	GUtilities::MarkLocation(location, index_count, in_size);


	GUtilities::InclusiveScan(location, *out_size);

	ulong *local_offset;

	checkCudaErrors(hipMalloc(&local_offset, *out_size * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, *out_size * sizeof(RESULT)));

	GUtilities::ComputeOffset(index_count, location, local_offset, *out_size);

	decompose(*out_bound, in_bound, location, local_offset, *out_size);

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(location));
}

void GPUHJ::Rebalance2(ulong *index_count, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size, hipStream_t stream)
{
	ExclusiveScanAsyncWrapper(index_count, in_size, out_size, stream);

	if (*out_size == 0) {
		return;
	}

	ulong *location;

	checkCudaErrors(hipMalloc(&location, sizeof(ulong) * (*out_size)));
	checkCudaErrors(hipMemsetAsync(location, 0, sizeof(ulong) * (*out_size), stream));

	GUtilities::MarkLocation(location, index_count, in_size, stream);

	GUtilities::InclusiveScan(location, *out_size, stream);

	ulong *local_offset;

	checkCudaErrors(hipMalloc(&local_offset, *out_size * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, *out_size * sizeof(RESULT)));

	GUtilities::ComputeOffset(index_count, location, local_offset, *out_size, stream);

	decompose(*out_bound, in_bound, location, local_offset, *out_size, stream);

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(location));
}

void GPUHJ::Rebalance(ulong *index_count, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size)
{

	int block_x, grid_x;

	block_x = (in_size < BLOCK_SIZE_X) ? in_size : BLOCK_SIZE_X;
	grid_x = (in_size - 1)/block_x + 1;

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	ulong *mark;
	ulong size_no_zeros;
	ResBound *tmp_bound;
	ulong sum;

	/* Remove zeros elements */
	ulong *no_zeros;

	checkCudaErrors(hipMalloc(&mark, (in_size + 1) * sizeof(ulong)));

	GUtilities::MarkNonZeros(index_count, in_size, mark);

	GUtilities::ExclusiveScan(mark, in_size + 1, &size_no_zeros);

	if (size_no_zeros == 0) {
		*out_size = 0;
		checkCudaErrors(hipFree(mark));

		return;
	}

	checkCudaErrors(hipMalloc(&no_zeros, (size_no_zeros + 1) * sizeof(ulong)));
	checkCudaErrors(hipMalloc(&tmp_bound, size_no_zeros * sizeof(ResBound)));

	GUtilities::RemoveZeros(index_count, in_bound, no_zeros, tmp_bound, mark, in_size);

	GUtilities::ExclusiveScan(no_zeros, size_no_zeros + 1, &sum);

	if (sum == 0) {
		*out_size = 0;
		checkCudaErrors(hipFree(mark));
		checkCudaErrors(hipFree(no_zeros));
		checkCudaErrors(hipFree(tmp_bound));

		return;
	}

	ulong *tmp_location, *local_offset;

	checkCudaErrors(hipMalloc(&tmp_location, sum * sizeof(ulong)));
	checkCudaErrors(hipMemset(tmp_location, 0, sizeof(ulong) * sum));
	checkCudaErrors(hipDeviceSynchronize());

	GUtilities::MarkTmpLocation(tmp_location, no_zeros, size_no_zeros);

	GUtilities::InclusiveScan(tmp_location, sum);

	checkCudaErrors(hipMalloc(&local_offset, sum * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, sum * sizeof(RESULT)));

	GUtilities::ComputeOffset(no_zeros, tmp_location, local_offset, sum);
	decompose(*out_bound, tmp_bound, tmp_location, local_offset, sum);

	*out_size = sum;

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(tmp_location));
	checkCudaErrors(hipFree(no_zeros));
	checkCudaErrors(hipFree(mark));
	checkCudaErrors(hipFree(tmp_bound));

}

void GPUHJ::Rebalance(ulong *index_count, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size, hipStream_t stream)
{

	int block_x, grid_x;

	block_x = (in_size < BLOCK_SIZE_X) ? in_size : BLOCK_SIZE_X;
	grid_x = (in_size - 1)/block_x + 1;

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	ulong *mark;
	ulong size_no_zeros;
	ResBound *tmp_bound;
	ulong sum;

	/* Remove zeros elements */
	ulong *no_zeros;

	checkCudaErrors(hipMalloc(&mark, (in_size + 1) * sizeof(ulong)));

	GUtilities::MarkNonZeros(index_count, in_size, mark, stream);

	GUtilities::ExclusiveScan(mark, in_size + 1, &size_no_zeros, stream);

	if (size_no_zeros == 0) {
		*out_size = 0;
		checkCudaErrors(hipFree(mark));

		return;
	}

	checkCudaErrors(hipMalloc(&no_zeros, (size_no_zeros + 1) * sizeof(ulong)));
	checkCudaErrors(hipMalloc(&tmp_bound, size_no_zeros * sizeof(ResBound)));

	GUtilities::RemoveZeros(index_count, in_bound, no_zeros, tmp_bound, mark, in_size, stream);

	GUtilities::ExclusiveScan(no_zeros, size_no_zeros + 1, &sum, stream);

	if (sum == 0) {
		*out_size = 0;
		checkCudaErrors(hipFree(mark));
		checkCudaErrors(hipFree(no_zeros));
		checkCudaErrors(hipFree(tmp_bound));

		return;
	}

	ulong *tmp_location, *local_offset;

	checkCudaErrors(hipMalloc(&tmp_location, sum * sizeof(ulong)));
	checkCudaErrors(hipMemset(tmp_location, 0, sizeof(ulong) * sum));
	checkCudaErrors(hipDeviceSynchronize());

	GUtilities::MarkTmpLocation(tmp_location, no_zeros, size_no_zeros, stream);

	GUtilities::InclusiveScan(tmp_location, sum, stream);

	checkCudaErrors(hipMalloc(&local_offset, sum * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, sum * sizeof(RESULT)));

	GUtilities::ComputeOffset(no_zeros, tmp_location, local_offset, sum, stream);
	decompose(*out_bound, tmp_bound, tmp_location, local_offset, sum, stream);

	*out_size = sum;

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(tmp_location));
	checkCudaErrors(hipFree(no_zeros));
	checkCudaErrors(hipFree(mark));
	checkCudaErrors(hipFree(tmp_bound));

}
}
